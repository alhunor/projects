#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <Cuda.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}







#define MAX 100

// this GPU kernel function calculates a random number and stores it in the parameter
__global__ void random(int* result, int nb)
{
  // keeps track of the seed value we will store a random state for every thread
  hiprandState_t state;
  const int seed = 123; /// the seed controls the sequence of random values that are produced
  // initialize the state
  hiprand_init(seed, 
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);

  // hiprand works like rand - except that it takes a state as a parameter
  for (int i = 0; i < nb; ++i)
  {
	  result[i] = hiprand(&state) % MAX;
  }
}

void mainRan(int nb)
{
  // allocate an int on the GPU 
  int* gpu_x;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventRecord(start, 0);
  hipEventCreate(&stop);

  hipMalloc((void**)&gpu_x, nb*sizeof(int));

  // invoke the GPU to initialize all of the random states
  random << <1, 1 >> >(gpu_x, nb);

  // copy the random number back
  int* x = (int*)malloc(sizeof(int)*nb);
  hipMemcpy(x, gpu_x, nb*sizeof(int), hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float time;
  hipEventElapsedTime(&time, start, stop);
  cout << "Elapsed:" << time<<endl;


  for (int i = 0; i < nb; ++i)
  {
	  printf("%d\n", x[i]);
  }

  // free the memory we allocated 
  hipFree(gpu_x);
  free(x);
}

















int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);

	_getch();

	mainRan(10);

	_getch();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
