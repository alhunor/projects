#include "hip/hip_runtime.h"
// http://stackoverflow.com/questions/14827160/how-do-i-enable-syntax-highlighting-of-cuda-cu-files-in-visual-studio-2010
 
#define WIN32
#include <stdio.h>
#include <stdlib.h>

//#include <windows.h>
//#include <iostream>
 
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include <dSFMT\dSFMT.h>
#include "hftimer.h"
 
//hipSuccess
//HIPRAND_STATUS_SUCCESS
class Curand
{
public:
        Curand(): initialised(false) {}
        ~ Curand() {if (initialised) destroy();}
        void init(int seed, hiprandRngType_t type=HIPRAND_RNG_PSEUDO_MTGP32, int _size=16384*4);
        float rand();
        // HIPRAND_RNG_PSEUDO_MRG32K3A - in double precision only
        // HIPRAND_RNG_PSEUDO_DEFAULT,
        // HIPRAND_RNG_PSEUDO_XORWOW, HIPRAND_RNG_PSEUDO_MTGP32
        // HIPRAND_RNG_QUASI_SOBOL32
protected:
        bool initialised;
        void destroy();
        hiprandGenerator_t gen;
        float* devPtr, *hostPtr;
        int size, pos;
};
 
char* curandGetErrorString(hiprandStatus_t status)
{
        switch ( (int)status)
        {
        case HIPRAND_STATUS_SUCCESS: //0
                return "No errors";
        case HIPRAND_STATUS_VERSION_MISMATCH : //100
                return "Header file and linked library version do not match";
        case HIPRAND_STATUS_NOT_INITIALIZED: //101
                return "Generator not initialized";
        case HIPRAND_STATUS_ALLOCATION_FAILED: //102
                return "Memory allocation failed";
        case HIPRAND_STATUS_TYPE_ERROR: //103
                return "Generator is wrong type";
        case HIPRAND_STATUS_OUT_OF_RANGE: //104
                return "Argument out of range";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE: //105
                return "Length requested is not a multple of dimension";
        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: //106
                return "GPU does not have double precision required by MRG32k3a";
        case HIPRAND_STATUS_LAUNCH_FAILURE: //201
                return "Kernel launch failure";
        case HIPRAND_STATUS_PREEXISTING_FAILURE : //202
                return "Preexisting failure on library entry";
        case HIPRAND_STATUS_INITIALIZATION_FAILED: // 203
                return "Initialization of CUDA failed";
        case HIPRAND_STATUS_ARCH_MISMATCH: //204
                return "Architecture mismatch, GPU does not support requested feature";
        default:
        case HIPRAND_STATUS_INTERNAL_ERROR : // 999
                return "Internal library error";
        } //switch
} // char* curandGetErrorString(hiprandStatus_t status)
 
void Curand::init(int seed, hiprandRngType_t type, int _size)
{
        hiprandStatus_t status;
        status = hiprandCreateGenerator(&gen, type);
        if (status!=HIPRAND_STATUS_SUCCESS)
        {
                printf("%s - Error in %s at line %d \n", curandGetErrorString(status), __FILE__, __LINE__ );
                exit(-1);
        }
        status = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
       
        size = _size;
        hipMalloc((void **)&devPtr, size*sizeof(float));
        hostPtr=new float[size];
        pos=size; // no numbers are available
        initialised=true;
} // Curand::init(int seed, hiprandRngType_t type, int _size)
 
float Curand::rand()
{
        if (pos<size)
                return hostPtr[pos++];
        hiprandGenerateUniform(gen,devPtr, size);
        hipMemcpy(hostPtr, devPtr, size*sizeof(float), hipMemcpyDeviceToHost);
        pos=1;
        return hostPtr[0];
} // float hiprand::rand()
 
void Curand::destroy()
{
        if (!initialised)
                return;
 
        hiprandDestroyGenerator(gen);
        delete[] hostPtr;
        hipFree(devPtr);
} // hiprand::destroy()
 
 
__global__ void init_stuff(hiprandState *state)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(123, idx, 0, &state[idx]);
}
 
__global__ void make_rand(hiprandState *state, float *randArray, int nRuns)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        float a, b;
        float count = 0;
        int i;
//      count = threadIdx.x;
 
        for (i=0; i<nRuns; i++)
        {
                a = hiprand_uniform(&state[idx]);
                b = hiprand_uniform(&state[idx]);
                if (a*a + b*b <=1) count++;
        }
 
        randArray[idx] = count;
}
 
 
// gets info about the <deviceNR>th CUDA device
void cudaDeviceInfo(int deviceNR, hipDeviceProp_t& deviceProp)
{
        int deviceCount = 0;
        int driverVersion = 0, runtimeVersion = 0;
 
        hipError_t error_id = hipGetDeviceCount(&deviceCount);
 
    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }
        if (deviceNR>deviceCount)
        {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);    
        }
 
        hipSetDevice(deviceNR);
        hipGetDeviceProperties(&deviceProp, deviceNR);
        printf("\nDevice %d: \"%s\"\n", deviceNR, deviceProp.name);
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
 
        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
    char msg[256];
    sprintf(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
            (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
    printf("%s", msg);
 
    printf("  (%2d) Multiprocessors x (%3d) CUDA Cores/MP:    %d CUDA Cores\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
 
    printf("  GPU Clock rate:                                %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
 
    printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n", deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
           deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
           deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
} // cudaDeviceInfo(int deviceNR)
 
const int nThreads=512;
const int nBlocks= 1;
const double N = nBlocks * nThreads * 200000;
 
double computePiOnHost()
{
        dsfmt_t dsfmt;
        double x,y;
        double count=0;
        int i;
       
        dsfmt_init_gen_rand(&dsfmt, 123);
 
        for (i=0; i<N; i++)
        {
                x = dsfmt_genrand_close_open(&dsfmt);
                y = dsfmt_genrand_close_open(&dsfmt);
                if (x*x +y*y<=1) count++;
        }
        return 4*count/N;
}
 
double computePiOnHost2()
{
        double x,y;
        double count=0;
        int i;
       
        Curand c;
        c.init(123);
 
        for (i=0; i<N; i++)
        {
                x = c.rand();
                y = c.rand();
                if (x*x +y*y<=1) count++;
        }
        return 4*count/N;
}
 
double computePiOnGPU()
{
        hiprandState *d_state;
        hipMalloc(&d_state, nThreads * nBlocks * sizeof(hiprandState) );
 
        float* randArray;
    hipMalloc((void**) &randArray, sizeof(float)* nBlocks*nThreads);
 
    // setup seeds
        init_stuff<<<nBlocks, nThreads>>>(d_state);
 
        int nRuns = (int)N/(nBlocks * nThreads);
        double count = 0;
    float host[nThreads * nBlocks];
 
        int j;
 
        make_rand<<<nBlocks, nThreads>>>(d_state, randArray, nRuns);
    hipMemcpy(host, randArray, sizeof(float)*nThreads * nBlocks, hipMemcpyDeviceToHost);
        for (j=0; j<nThreads * nBlocks; j++) count += host[j];
 
        hipFree(d_state);
        hipFree(randArray);
 
        return 4*count/N;
} // double computePiOnGPU()
 
 
__global__ void computePiOnGPU2_kernel(hiprandStateMtgp32 * devState, float sampleCount, float* devArray)
{
        int id = threadIdx.x + blockIdx.x*256;
        float count=0;
        int i;
        float x,y;
        //hiprandStateMtgp32 localState = devState[blockIdx.x];
        for (i=0; i<sampleCount; ++i)
        {
                x = hiprand_uniform(&devState[blockIdx.x]);
                y = hiprand_uniform(&devState[blockIdx.x]);
                //x = hiprand_uniform(&localState);
                //y = hiprand_uniform(&localState);
                if (x*x+y*y<=1) count++;
        }
        //devState[blockIdx.x] = localState;
        devArray[id] += count/sampleCount;
} // computePiOnGPU2_kernel
 
 
double computePiOnGPU2()
{       // using MTGP32
       
    int i;
    double total=0;
    hiprandStateMtgp32 * devState;
    mtgp32_kernel_params * devKernelParams;
    float* devArray, hostArray[64*256];
    float sampleCount = 100;
    const int nbRuns = 100;

    hipMalloc((void**)&devState, 64 * sizeof(hiprandStateMtgp32) );
    hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params) );
    hipMalloc((void**) &devArray, sizeof(float)*64*256);
 
    hipMemset(devArray, 0, 64*256*sizeof(float));

    int status = (int ) hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams);
    status = (int) hiprandMakeMTGP32KernelState(devState, mtgp32dc_params_fast_11213, devKernelParams, 64, 1234);

    for (i=0; i<nbRuns; i++)
    {
            computePiOnGPU2_kernel<<<64, 256>>>(devState, sampleCount, devArray);
    }
 
    hipMemcpy(hostArray, devArray, sizeof(float)*64*256, hipMemcpyDeviceToHost);

    hipFree(devState);
    hipFree(devArray);

    for (i=0; i<64*256; ++i)
    {
            total += hostArray[i];
    }
    total=(total*4)/(256*64*nbRuns);

    return total;
} // double computePiOnGPU2()
 
 
int main()
{
	hftimer timer; 
	hipDeviceProp_t deviceProp;

    // first device starts at 0.
    // Get CUDA device capabilities
    cudaDeviceInfo(0, deviceProp);
   
     printf("Reference value of PI is 3.1415 9265 3589 793 23846 26433 \n");
    long t;
 
	timer.startTimer();
 //   printf("Host PI=%f\n",computePiOnHost2() );
	t = timer.getTime();
	timer.startTimer();
//    printf("Host PI=%f\n",computePiOnHost() );
	printf("GPU PI=%f\n",computePiOnGPU2() );
	t = timer.getTime();
	 
	return 0;
} // int main()